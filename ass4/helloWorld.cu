
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(int* a, int* b, int* c)
{
	*c = *a + *b;
}


int main()
{
	int a, b, c;
	int *d_a, *d_b, *d_c;
	a = 7;
	b = 8;
	hipMalloc((void **)&d_a, sizeof(int));
	hipMalloc((void **)&d_b, sizeof(int));
	hipMalloc((void **)&d_c, sizeof(int));
	hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
	kernel<<<1, 1>>>(d_a, d_b, d_c);
	hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("Asd %d\n", c);
	return 0;
}