#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void kernel(int* a, int* b, int* c)
{
	*c = *a + *b;
}


int main()
{
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	*a = 7;
	*b = 8;
	d_a = hipMalloc(sizeof(int));
	d_b = hipMalloc(sizeof(int));
	hipMemcpy(d_a, a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(int), hipMemcpyHostToDevice);
	kernel<<<1, 1>>>(d_a, d_b, d_c);
	hipMemcpy(d_c, c, sizeof(int), hipMemcpyDeviceToHost);
	printf("Asd %d\n", *d_c);
	return 0;
}