#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void kernel(int* a, int* b, int* c)
{
	*c = *a + *b;
}


int main()
{
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	*a = 7;
	*b = 8;
	d_a = hipMalloc((void **)&d_a, sizeof(int));
	d_b = hipMalloc((void **)&d_b, sizeof(int));
	hipMemcpy(d_a, a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(int), hipMemcpyHostToDevice);
	kernel<<<1, 1>>>(d_a, d_b, d_c);
	hipMemcpy(c, d_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("Asd %d\n", *c);
	return 0;
}